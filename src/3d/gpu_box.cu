#include "hip/hip_runtime.h"
#include <mma.h>
#include <iostream>
#include "3d_utils.h"
#include <chrono>

using namespace nvcuda;

#define BLOCK_SIZE_ROW 8
#define BLOCK_SIZE_COL 64
#define HALO 3
#define UNIT_LENGTH 7
#define D_BLOCK_SIZE_COL (BLOCK_SIZE_COL + HALO * 2)
#define D_BLOCK_SIZE_ROW (BLOCK_SIZE_ROW + HALO * 2)
#define PAD 2
#define SM_SIZE_COL (UNIT_LENGTH * D_BLOCK_SIZE_ROW + PAD)
#define SM_SIZE_ROW (D_BLOCK_SIZE_COL / (UNIT_LENGTH + 1))
#define WARP_PER_BLOCK 8
#define COL_PER_WARP (BLOCK_SIZE_ROW / WARP_PER_BLOCK * UNIT_LENGTH)
#define TENSOR_CORE_M 8
#define MMA_NUM 13
#define IDX2D(x, y, ldm) ((x) * (ldm) + (y))
#define IDX3D(x, y, z, rows, cols) ((x) * (rows) * (cols) + (y) * (cols) + (z))

__constant__ double param_matrix_d[4 * 2 * 52 * TENSOR_CORE_M];

__forceinline__ __device__ void load_shared_data(double * __restrict__ data, const double * __restrict__ in, const int h, const int rows, const int cols, const int * __restrict__ lookup_table1, const int * __restrict__ lookup_table2) {
    int begin = IDX3D(h, blockIdx.x * BLOCK_SIZE_ROW, blockIdx.y * BLOCK_SIZE_COL, rows, cols);
    int tid = threadIdx.x;
    int total_threads = blockDim.x;
    for (int i = tid; i < D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL; i += total_threads) {
        int row = i / D_BLOCK_SIZE_COL;
        int col = i % D_BLOCK_SIZE_COL;
        data[IDX2D(0, lookup_table1[i], SM_SIZE_ROW * SM_SIZE_COL)] = in[begin + IDX2D(row, col, cols)];
        data[IDX2D(1, lookup_table2[i], SM_SIZE_ROW * SM_SIZE_COL)] = in[begin + IDX2D(row, col, cols)];
    }
    __syncthreads();
}

__forceinline__ __device__ void load_shared_data_b4(double * __restrict__ data, const double * __restrict__ in, const int h, const int rows, const int cols, const int * __restrict__ lookup_table1, const int * __restrict__ lookup_table2) {
    int begin = IDX3D(h, blockIdx.x * BLOCK_SIZE_ROW, blockIdx.y * BLOCK_SIZE_COL, rows, cols);
    int tid = threadIdx.x;
    int total_threads = blockDim.x;
    for (int i = tid; i < D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL; i += total_threads) {
        int row = i / D_BLOCK_SIZE_COL;
        int col = i % D_BLOCK_SIZE_COL;
        if (lookup_table1[i] != -1)
            data[IDX2D(0, lookup_table1[i], SM_SIZE_ROW * SM_SIZE_COL)] = in[begin + IDX2D(row, col, cols)];
        if (lookup_table2[i] != -1)
            data[IDX2D(1, lookup_table2[i], SM_SIZE_ROW * SM_SIZE_COL)] = in[begin + IDX2D(row, col, cols)];
    }
    __syncthreads();
}

__forceinline__ __device__ void load_shared_data_b3(double * __restrict__ data, const double * __restrict__ in, const int h, const int rows, const int cols, const int * __restrict__ lookup_table1, const int * __restrict__ lookup_table2) {
    int begin = IDX3D(h, blockIdx.x * BLOCK_SIZE_ROW, blockIdx.y * BLOCK_SIZE_COL, rows, cols);
    int tid = threadIdx.x;
    int total_threads = blockDim.x;
    for (int i = tid; i < D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL; i += total_threads) {
        int row = i / D_BLOCK_SIZE_COL;
        int col = i % D_BLOCK_SIZE_COL;
        if (lookup_table1[i] != -1)
            data[IDX2D(0, lookup_table1[i], SM_SIZE_ROW * (SM_SIZE_COL - PAD))] = in[begin + IDX2D(row, col, cols)];
        if (lookup_table2[i] != -1)
            data[IDX2D(1, lookup_table2[i], SM_SIZE_ROW * (SM_SIZE_COL - PAD))] = in[begin + IDX2D(row, col, cols)];
    }
    __syncthreads();
}

__forceinline__ __device__ void load_shared_data_b1(double * out, double * __restrict__ data, const double * __restrict__ in, const int h, const int rows, const int cols, const int * __restrict__ lookup_table1, const int * __restrict__ lookup_table2) {
    int begin = IDX3D(h, blockIdx.x * BLOCK_SIZE_ROW, blockIdx.y * BLOCK_SIZE_COL, rows, cols);
    int tid = threadIdx.x;
    int total_threads = blockDim.x;
    for (int i = tid; i < D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL; i += total_threads) {
        int row = i / D_BLOCK_SIZE_COL;
        int col = i % D_BLOCK_SIZE_COL;
        if (lookup_table1[i] != -1) {
            data[IDX2D(0, lookup_table1[i], SM_SIZE_ROW * (SM_SIZE_COL - PAD))] = in[begin + IDX2D(row, col, cols)];
            out[IDX2D(0, lookup_table1[i], SM_SIZE_ROW * (SM_SIZE_COL - PAD))] = in[begin + IDX2D(row, col, cols)];
            data[IDX2D(0, lookup_table1[i], SM_SIZE_ROW * (SM_SIZE_COL - PAD))] = out[IDX2D(0, lookup_table1[i], SM_SIZE_ROW * (SM_SIZE_COL))];
        }
        if (lookup_table2[i] != -1) {
            data[IDX2D(1, lookup_table2[i], SM_SIZE_ROW * (SM_SIZE_COL - PAD))] = in[begin + IDX2D(row, col, cols)];
            out[IDX2D(1, lookup_table2[i], SM_SIZE_ROW * (SM_SIZE_COL - PAD))] = in[begin + IDX2D(row, col, cols)];
            data[IDX2D(1, lookup_table2[i], SM_SIZE_ROW * (SM_SIZE_COL - PAD))] = out[IDX2D(1, lookup_table2[i], SM_SIZE_ROW * (SM_SIZE_COL))];
        }
    }
    __syncthreads();
}

__forceinline__ __device__ void compute_tensorcore(double * __restrict__ data, double * __restrict__ out, const int ldm, const int warp_id, const int param_idx) {
    wmma::fragment<wmma::matrix_b, 8, 8, 4, double, wmma::row_major> param_frag[2][MMA_NUM];
#pragma unroll
    for (int i = 0; i < MMA_NUM; i++) {
        wmma::load_matrix_sync(param_frag[0][i], param_matrix_d + param_idx * 2 * 52 * TENSOR_CORE_M + i * 32, 8);
        wmma::load_matrix_sync(param_frag[1][i], param_matrix_d + param_idx * 2 * 52 * TENSOR_CORE_M + 52 * 8 + i * 32, 8);
    }

    wmma::fragment<wmma::accumulator, 8, 8, 4, double> acc_frag;
    wmma::fragment<wmma::matrix_a, 8, 8, 4, double, wmma::row_major> in_frag;

    for (int col = warp_id * COL_PER_WARP; col < warp_id * COL_PER_WARP + COL_PER_WARP; col += UNIT_LENGTH) {
        wmma::fill_fragment(acc_frag, 0.0);
#pragma unroll
        for (int compute_idx = 0; compute_idx < MMA_NUM; compute_idx++) {
            wmma::load_matrix_sync(in_frag, data + IDX2D(0, col + compute_idx * 4, SM_SIZE_COL), SM_SIZE_COL);
            wmma::mma_sync(acc_frag, in_frag, param_frag[0][compute_idx], acc_frag);
        }
#pragma unroll
        for (int compute_idx = 0; compute_idx < MMA_NUM; compute_idx++) {
            wmma::load_matrix_sync(in_frag, data + SM_SIZE_ROW * SM_SIZE_COL + IDX2D(0, col + compute_idx * 4, SM_SIZE_COL), SM_SIZE_COL);
            wmma::mma_sync(acc_frag, in_frag, param_frag[1][compute_idx], acc_frag);
        }
        wmma::store_matrix_sync(out + IDX2D(col / UNIT_LENGTH, 0, BLOCK_SIZE_COL), acc_frag, TENSOR_CORE_M, wmma::mem_row_major);
    }

    __syncthreads();
}

__forceinline__ __device__ void compute_tensorcore_b3(double * __restrict__ data, double * __restrict__ out, const int ldm, const int warp_id, const int param_idx) {
    wmma::fragment<wmma::matrix_b, 8, 8, 4, double, wmma::row_major> param_frag[2][MMA_NUM];
#pragma unroll
    for (int i = 0; i < MMA_NUM; i++) {
        wmma::load_matrix_sync(param_frag[0][i], param_matrix_d + param_idx * 2 * 52 * TENSOR_CORE_M + i * 32, 8);
        wmma::load_matrix_sync(param_frag[1][i], param_matrix_d + param_idx * 2 * 52 * TENSOR_CORE_M + 52 * 8 + i * 32, 8);
    }

    wmma::fragment<wmma::accumulator, 8, 8, 4, double> acc_frag;
    wmma::fragment<wmma::matrix_a, 8, 8, 4, double, wmma::row_major> in_frag;

    for (int col = warp_id * COL_PER_WARP; col < warp_id * COL_PER_WARP + COL_PER_WARP; col += UNIT_LENGTH) {
        wmma::fill_fragment(acc_frag, 0.0);
#pragma unroll
        for (int compute_idx = 0; compute_idx < MMA_NUM; compute_idx++) {
            wmma::load_matrix_sync(in_frag, data + IDX2D(0, col + compute_idx * 4, (SM_SIZE_COL - PAD)), (SM_SIZE_COL - PAD));
            wmma::mma_sync(acc_frag, in_frag, param_frag[0][compute_idx], acc_frag);
        }
#pragma unroll
        for (int compute_idx = 0; compute_idx < MMA_NUM; compute_idx++) {
            wmma::load_matrix_sync(in_frag, data + SM_SIZE_ROW * (SM_SIZE_COL - PAD) + IDX2D(0, col + compute_idx * 4, (SM_SIZE_COL - PAD)), (SM_SIZE_COL - PAD));
            wmma::mma_sync(acc_frag, in_frag, param_frag[1][compute_idx], acc_frag);
        }
        wmma::store_matrix_sync(out + IDX2D(col / UNIT_LENGTH, 0, BLOCK_SIZE_COL), acc_frag, TENSOR_CORE_M, wmma::mem_row_major);
    }

    __syncthreads();
}

__forceinline__ __device__ void compute_tensorcore_b2(double * __restrict__ data, double * __restrict__ out, const int ldm, const int warp_id, const int param_idx) {
    int thread_id = threadIdx.x % 32;
    int x = thread_id / 4;
    int y = thread_id % 4;
    for (int col = warp_id * COL_PER_WARP; col < warp_id * COL_PER_WARP + COL_PER_WARP; col += UNIT_LENGTH) {
        double sum = 0.0;
        double sum1 = 0.0;
#pragma unroll
        for (int compute_idx = 0; compute_idx < MMA_NUM; compute_idx++) {
            for (int z = 0; z < 6; z++) {
                sum += *(data + IDX2D(0 + x / 4, col + compute_idx * 4 + z, (SM_SIZE_COL - PAD))) * param_matrix_d[z * 8 + y];
                sum1 += *(data + IDX2D(0 + x / 4, col + compute_idx * 4 + z, (SM_SIZE_COL - PAD))) * param_matrix_d[z * 8 + y + 1];
            }
        }
#pragma unroll
        for (int compute_idx = 0; compute_idx < MMA_NUM; compute_idx++) {
            for (int z = 0; z < 6; z++) {
                sum += *(data + SM_SIZE_ROW * (SM_SIZE_COL - PAD) + IDX2D(0 + x / 4, col + compute_idx * 4 + z, (SM_SIZE_COL - PAD))) * param_matrix_d[z * 8 + y];
                sum1 += *(data + SM_SIZE_ROW * (SM_SIZE_COL - PAD) + IDX2D(0 + x / 4, col + compute_idx * 4 + z, (SM_SIZE_COL - PAD))) * param_matrix_d[z * 8 + y + 1];
            }
        }
        out[IDX2D(col / UNIT_LENGTH, 0, BLOCK_SIZE_COL)] = sum;
        out[IDX2D(col / UNIT_LENGTH, 0, BLOCK_SIZE_COL) + 1] = sum1;
    }

    __syncthreads();
}

__forceinline__ __device__ void add(double * __restrict__ data1, double * __restrict__ data2, double * __restrict__ data3, double * __restrict__ data4, double * __restrict__ data5, double * __restrict__ data6, double * __restrict__ data7, double * __restrict__ out, const int cols) {
    int tid = threadIdx.x;
    int total_threads = blockDim.x;
    for (int i = tid; i < BLOCK_SIZE_ROW * BLOCK_SIZE_COL; i += total_threads) {
        int row = i / BLOCK_SIZE_COL;
        int col = i % BLOCK_SIZE_COL;
        out[IDX2D(row, col, cols)] = data1[i] + data2[i] + data3[i] + data4[i] + data5[i] + data6[i] + data7[i];
    }
}

__global__ void gpu_box_3d1r_kernel (const double * __restrict__ in, double * __restrict__ out, const int heights, const int rows, const int cols, const int * __restrict__ lookup_table1, const int * __restrict__ lookup_table2) {
    extern __shared__ double data[];
    double * intermediate = &data[2 * SM_SIZE_ROW * SM_SIZE_COL];
    // __shared__ double data[2][SM_SIZE_ROW * SM_SIZE_COL];
    // __shared__ double intermediate[19][BLOCK_SIZE_ROW * BLOCK_SIZE_COL];

    int begin =  IDX2D(blockIdx.x * BLOCK_SIZE_ROW, blockIdx.y * BLOCK_SIZE_COL, cols);
    int warp_idx = threadIdx.x / 32;
    // int tid = threadIdx.x;
    // int total_threads = blockDim.x;

    load_shared_data(data, in, 0, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore(data, intermediate, SM_SIZE_COL, warp_idx, 0);
    load_shared_data(data, in, 1, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore(data, intermediate + BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 0);
    compute_tensorcore(data, intermediate + 7 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 1);
    load_shared_data(data, in, 2, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore(data, intermediate + 2 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 0);
    compute_tensorcore(data, intermediate + 8 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 1);
    compute_tensorcore(data, intermediate + 12 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 2);
    load_shared_data(data, in, 3, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore(data, intermediate + 3 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 0);
    compute_tensorcore(data, intermediate + 9 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 1);
    compute_tensorcore(data, intermediate + 13 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 2);
    compute_tensorcore(data, intermediate + 16 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 3);
    load_shared_data(data, in, 4, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore(data, intermediate + 4 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 0);
    compute_tensorcore(data, intermediate + 10 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 1);
    compute_tensorcore(data, intermediate + 14 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 2);
    compute_tensorcore(data, intermediate + 17 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 3);
    load_shared_data(data, in, 5, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore(data, intermediate + 5 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 0);
    compute_tensorcore(data, intermediate + 11 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 1);
    compute_tensorcore(data, intermediate + 15 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 2);
    compute_tensorcore(data, intermediate + 18 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 3);
    for (int h = 6; h < heights + 6; h++) {
        load_shared_data(data, in, h, rows, cols, lookup_table1, lookup_table2);
        compute_tensorcore(data, intermediate + (h % 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 0);
        add(
            intermediate + ((h - 6) % 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 6) % 5 + 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 6) % 4 + 12) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 6) % 3 + 16) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 4) % 4 + 12) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 2) % 5 + 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + (h % 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            out + (h - 3) * rows * cols + begin + IDX2D(HALO, HALO, cols),
            cols);
        compute_tensorcore(data, intermediate + ((h - 6) % 5 + 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 1);
        compute_tensorcore(data, intermediate + ((h - 6) % 4 + 12) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 2);
        compute_tensorcore(data, intermediate + ((h - 6) % 3 + 16) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 3);
    }
}

__global__ void breakdown4_kernel (const double * __restrict__ in, double * __restrict__ out, const int heights, const int rows, const int cols, const int * __restrict__ lookup_table1, const int * __restrict__ lookup_table2) {
    extern __shared__ double data[];
    double * intermediate = &data[2 * SM_SIZE_ROW * SM_SIZE_COL];
    // __shared__ double data[2][SM_SIZE_ROW * SM_SIZE_COL];
    // __shared__ double intermediate[19][BLOCK_SIZE_ROW * BLOCK_SIZE_COL];

    int begin =  IDX2D(blockIdx.x * BLOCK_SIZE_ROW, blockIdx.y * BLOCK_SIZE_COL, cols);
    int warp_idx = threadIdx.x / 32;
    // int tid = threadIdx.x;
    // int total_threads = blockDim.x;

    load_shared_data_b4(data, in, 0, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore(data, intermediate, SM_SIZE_COL, warp_idx, 0);
    load_shared_data_b4(data, in, 1, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore(data, intermediate + BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 0);
    compute_tensorcore(data, intermediate + 7 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 1);
    load_shared_data_b4(data, in, 2, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore(data, intermediate + 2 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 0);
    compute_tensorcore(data, intermediate + 8 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 1);
    compute_tensorcore(data, intermediate + 12 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 2);
    load_shared_data_b4(data, in, 3, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore(data, intermediate + 3 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 0);
    compute_tensorcore(data, intermediate + 9 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 1);
    compute_tensorcore(data, intermediate + 13 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 2);
    compute_tensorcore(data, intermediate + 16 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 3);
    load_shared_data_b4(data, in, 4, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore(data, intermediate + 4 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 0);
    compute_tensorcore(data, intermediate + 10 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 1);
    compute_tensorcore(data, intermediate + 14 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 2);
    compute_tensorcore(data, intermediate + 17 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 3);
    load_shared_data_b4(data, in, 5, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore(data, intermediate + 5 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 0);
    compute_tensorcore(data, intermediate + 11 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 1);
    compute_tensorcore(data, intermediate + 15 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 2);
    compute_tensorcore(data, intermediate + 18 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 3);
    for (int h = 6; h < heights + 6; h++) {
        load_shared_data_b4(data, in, h, rows, cols, lookup_table1, lookup_table2);
        compute_tensorcore(data, intermediate + (h % 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 0);
        add(
            intermediate + ((h - 6) % 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 6) % 5 + 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 6) % 4 + 12) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 6) % 3 + 16) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 4) % 4 + 12) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 2) % 5 + 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + (h % 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            out + (h - 3) * rows * cols + begin + IDX2D(HALO, HALO, cols),
            cols);
        compute_tensorcore(data, intermediate + ((h - 6) % 5 + 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 1);
        compute_tensorcore(data, intermediate + ((h - 6) % 4 + 12) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 2);
        compute_tensorcore(data, intermediate + ((h - 6) % 3 + 16) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, SM_SIZE_COL, warp_idx, 3);
    }
}

__global__ void breakdown3_kernel (const double * __restrict__ in, double * __restrict__ out, const int heights, const int rows, const int cols, const int * __restrict__ lookup_table1, const int * __restrict__ lookup_table2) {
    extern __shared__ double data[];
    double * intermediate = &data[2 * SM_SIZE_ROW * (SM_SIZE_COL - PAD)];

    int begin =  IDX2D(blockIdx.x * BLOCK_SIZE_ROW, blockIdx.y * BLOCK_SIZE_COL, cols);
    int warp_idx = threadIdx.x / 32;

    load_shared_data_b3(data, in, 0, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore_b3(data, intermediate, (SM_SIZE_COL - PAD), warp_idx, 0);
    load_shared_data_b3(data, in, 1, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore_b3(data, intermediate + BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 0);
    compute_tensorcore_b3(data, intermediate + 7 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 1);
    load_shared_data_b3(data, in, 2, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore_b3(data, intermediate + 2 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 0);
    compute_tensorcore_b3(data, intermediate + 8 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 1);
    compute_tensorcore_b3(data, intermediate + 12 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 2);
    load_shared_data_b3(data, in, 3, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore_b3(data, intermediate + 3 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 0);
    compute_tensorcore_b3(data, intermediate + 9 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 1);
    compute_tensorcore_b3(data, intermediate + 13 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 2);
    compute_tensorcore_b3(data, intermediate + 16 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 3);
    load_shared_data_b3(data, in, 4, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore_b3(data, intermediate + 4 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 0);
    compute_tensorcore_b3(data, intermediate + 10 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 1);
    compute_tensorcore_b3(data, intermediate + 14 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 2);
    compute_tensorcore_b3(data, intermediate + 17 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 3);
    load_shared_data_b3(data, in, 5, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore_b3(data, intermediate + 5 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 0);
    compute_tensorcore_b3(data, intermediate + 11 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 1);
    compute_tensorcore_b3(data, intermediate + 15 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 2);
    compute_tensorcore_b3(data, intermediate + 18 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 3);
    for (int h = 6; h < heights + 6; h++) {
        load_shared_data_b3(data, in, h, rows, cols, lookup_table1, lookup_table2);
        compute_tensorcore_b3(data, intermediate + (h % 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 0);
        add(
            intermediate + ((h - 6) % 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 6) % 5 + 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 6) % 4 + 12) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 6) % 3 + 16) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 4) % 4 + 12) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 2) % 5 + 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + (h % 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            out + (h - 3) * rows * cols + begin + IDX2D(HALO, HALO, cols),
            cols);
        compute_tensorcore_b3(data, intermediate + ((h - 6) % 5 + 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 1);
        compute_tensorcore_b3(data, intermediate + ((h - 6) % 4 + 12) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 2);
        compute_tensorcore_b3(data, intermediate + ((h - 6) % 3 + 16) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 3);
    }
}

__global__ void breakdown2_kernel (const double * __restrict__ in, double * __restrict__ out, const int heights, const int rows, const int cols, const int * __restrict__ lookup_table1, const int * __restrict__ lookup_table2) {
    extern __shared__ double data[];
    double * intermediate = &data[2 * SM_SIZE_ROW * (SM_SIZE_COL - PAD)];

    int begin =  IDX2D(blockIdx.x * BLOCK_SIZE_ROW, blockIdx.y * BLOCK_SIZE_COL, cols);
    int warp_idx = threadIdx.x / 32;

    load_shared_data_b3(data, in, 0, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore_b2(data, intermediate, (SM_SIZE_COL - PAD), warp_idx, 0);
    load_shared_data_b3(data, in, 1, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore_b2(data, intermediate + BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 0);
    compute_tensorcore_b2(data, intermediate + 7 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 1);
    load_shared_data_b3(data, in, 2, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore_b2(data, intermediate + 2 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 0);
    compute_tensorcore_b2(data, intermediate + 8 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 1);
    compute_tensorcore_b2(data, intermediate + 12 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 2);
    load_shared_data_b3(data, in, 3, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore_b2(data, intermediate + 3 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 0);
    compute_tensorcore_b2(data, intermediate + 9 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 1);
    compute_tensorcore_b2(data, intermediate + 13 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 2);
    compute_tensorcore_b2(data, intermediate + 16 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 3);
    load_shared_data_b3(data, in, 4, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore_b2(data, intermediate + 4 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 0);
    compute_tensorcore_b2(data, intermediate + 10 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 1);
    compute_tensorcore_b2(data, intermediate + 14 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 2);
    compute_tensorcore_b2(data, intermediate + 17 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 3);
    load_shared_data_b3(data, in, 5, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore_b2(data, intermediate + 5 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 0);
    compute_tensorcore_b2(data, intermediate + 11 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 1);
    compute_tensorcore_b2(data, intermediate + 15 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 2);
    compute_tensorcore_b2(data, intermediate + 18 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 3);
    for (int h = 6; h < heights + 6; h++) {
        load_shared_data_b3(data, in, h, rows, cols, lookup_table1, lookup_table2);
        compute_tensorcore_b2(data, intermediate + (h % 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 0);
        add(
            intermediate + ((h - 6) % 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 6) % 5 + 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 6) % 4 + 12) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 6) % 3 + 16) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 4) % 4 + 12) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 2) % 5 + 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + (h % 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            out + (h - 3) * rows * cols + begin + IDX2D(HALO, HALO, cols),
            cols);
        compute_tensorcore_b2(data, intermediate + ((h - 6) % 5 + 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 1);
        compute_tensorcore_b2(data, intermediate + ((h - 6) % 4 + 12) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 2);
        compute_tensorcore_b2(data, intermediate + ((h - 6) % 3 + 16) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 3);
    }
}

__global__ void breakdown1_kernel (const double * __restrict__ in, double * __restrict__ out, const int heights, const int rows, const int cols, const int * __restrict__ lookup_table1, const int * __restrict__ lookup_table2) {
    extern __shared__ double data[];
    double * intermediate = &data[2 * SM_SIZE_ROW * (SM_SIZE_COL - PAD)];

    int begin =  IDX2D(blockIdx.x * BLOCK_SIZE_ROW, blockIdx.y * BLOCK_SIZE_COL, cols);
    int warp_idx = threadIdx.x / 32;

    load_shared_data_b1(out, data, in, 0, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore_b2(data, intermediate, (SM_SIZE_COL - PAD), warp_idx, 0);
    load_shared_data_b1(out, data, in, 1, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore_b2(data, intermediate + BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 0);
    compute_tensorcore_b2(data, intermediate + 7 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 1);
    load_shared_data_b1(out, data, in, 2, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore_b2(data, intermediate + 2 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 0);
    compute_tensorcore_b2(data, intermediate + 8 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 1);
    compute_tensorcore_b2(data, intermediate + 12 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 2);
    load_shared_data_b1(out, data, in, 3, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore_b2(data, intermediate + 3 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 0);
    compute_tensorcore_b2(data, intermediate + 9 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 1);
    compute_tensorcore_b2(data, intermediate + 13 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 2);
    compute_tensorcore_b2(data, intermediate + 16 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 3);
    load_shared_data_b1(out, data, in, 4, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore_b2(data, intermediate + 4 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 0);
    compute_tensorcore_b2(data, intermediate + 10 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 1);
    compute_tensorcore_b2(data, intermediate + 14 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 2);
    compute_tensorcore_b2(data, intermediate + 17 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 3);
    load_shared_data_b1(out, data, in, 5, rows, cols, lookup_table1, lookup_table2);
    compute_tensorcore_b2(data, intermediate + 5 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 0);
    compute_tensorcore_b2(data, intermediate + 11 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 1);
    compute_tensorcore_b2(data, intermediate + 15 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 2);
    compute_tensorcore_b2(data, intermediate + 18 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 3);
    for (int h = 6; h < heights + 6; h++) {
        load_shared_data_b1(out, data, in, h, rows, cols, lookup_table1, lookup_table2);
        compute_tensorcore_b2(data, intermediate + (h % 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 0);
        add(
            intermediate + ((h - 6) % 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 6) % 5 + 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 6) % 4 + 12) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 6) % 3 + 16) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 4) % 4 + 12) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + ((h - 2) % 5 + 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            intermediate + (h % 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, 
            out + (h - 3) * rows * cols + begin + IDX2D(HALO, HALO, cols),
            cols);
        compute_tensorcore_b2(data, intermediate + ((h - 6) % 5 + 7) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 1);
        compute_tensorcore_b2(data, intermediate + ((h - 6) % 4 + 12) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 2);
        compute_tensorcore_b2(data, intermediate + ((h - 6) % 3 + 16) * BLOCK_SIZE_ROW * BLOCK_SIZE_COL, (SM_SIZE_COL - PAD), warp_idx, 3);
    }
}



void gpu_box_3d1r(const double * __restrict__ in, double * __restrict__ out, const double * __restrict__ params, const int times, const int input_h, const int input_m, const int input_n) {
    double param_matrix_h[4][2][52 * 8] = {0.0};

    // Initialize parameter matrix
    for (int h = 0; h < 4; h++) {
        for (int col = 0; col < TENSOR_CORE_M; col++) {
            for(int i = 0; i < UNIT_LENGTH; i++) {
                for(int j = 0; j < UNIT_LENGTH; j++) {
                    if (j >= col) {
                        param_matrix_h[h][0][(i * UNIT_LENGTH + j) * 8 + col] = params[h * UNIT_LENGTH * UNIT_LENGTH + i * UNIT_LENGTH + j - col];
                    }
                }
            }
        }
        for (int col = 0; col < TENSOR_CORE_M; col++) {
            for(int i = 0; i < UNIT_LENGTH; i++) {
                for(int j = 0; j < UNIT_LENGTH; j++) {
                    if (j < col) {
                        param_matrix_h[h][1][(i * UNIT_LENGTH + j) * 8 + col] = params[h * UNIT_LENGTH * UNIT_LENGTH + i * UNIT_LENGTH + j - col + 7];
                    }
                }
            }
        }
    }

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(param_matrix_d), param_matrix_h, 4 * 2 * 8 * 52 * sizeof(double)));

    const int heights = input_h + 2 * HALO;
    const int rows = input_m + 2 * HALO;
    const int cols = input_n + 2 * HALO;
    const size_t array_size = heights * rows * cols * sizeof(double);
    double *array_d[2];
    CUDA_CHECK(hipMalloc(&array_d[0], array_size));
    CUDA_CHECK(hipMalloc(&array_d[1], array_size));
    CUDA_CHECK(hipMemset(array_d[0], 0, array_size));
    CUDA_CHECK(hipMemcpy(array_d[0], in, array_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(array_d[1], 0, array_size));

    const int BLOCK_M = (input_m + BLOCK_SIZE_ROW - 1) / BLOCK_SIZE_ROW; 
    const int BLOCK_N = (input_n + BLOCK_SIZE_COL - 1) / BLOCK_SIZE_COL; 
    dim3 grid_config(BLOCK_M, BLOCK_N);
    // dim3 grid_config(1, 1);
    dim3 block_config(32 * WARP_PER_BLOCK);
    int sm_size = (2 * SM_SIZE_ROW * SM_SIZE_COL + 19 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL) * sizeof(double);
    CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(gpu_box_3d1r_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, sm_size));

    int lookup_table1_h[D_BLOCK_SIZE_ROW][D_BLOCK_SIZE_COL];
    int lookup_table2_h[D_BLOCK_SIZE_ROW][D_BLOCK_SIZE_COL];
    for (int i = 0; i < D_BLOCK_SIZE_ROW; i++) {
        for (int j = 0; j < D_BLOCK_SIZE_COL; j++) {
            if ((j + 1) % 8 != 0 && j < D_BLOCK_SIZE_COL - 2 * HALO - 1) {
                lookup_table1_h[i][j] = IDX2D(j / (UNIT_LENGTH + 1), UNIT_LENGTH * i + j % (UNIT_LENGTH + 1), SM_SIZE_COL);
            } else {
                lookup_table1_h[i][j] = SM_SIZE_ROW * SM_SIZE_COL - 1;
            }
            if ((j + 2) % 8 != 0 && j > 2 * HALO) {
                lookup_table2_h[i][j] = IDX2D((j - UNIT_LENGTH) / (UNIT_LENGTH + 1), UNIT_LENGTH * i + (j - UNIT_LENGTH) % (UNIT_LENGTH + 1), SM_SIZE_COL);
            } else {
                lookup_table2_h[i][j] = SM_SIZE_ROW * SM_SIZE_COL - 1;
            }
        }
    }

    int * lookup_table1_d;
    int * lookup_table2_d;
    CUDA_CHECK(hipMalloc(&lookup_table1_d, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int)));
    CUDA_CHECK(hipMalloc(&lookup_table2_d, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int)));
    CUDA_CHECK(hipMemcpy(lookup_table1_d, lookup_table1_h, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(lookup_table2_d, lookup_table2_h, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int), hipMemcpyHostToDevice));

    int i = 0;

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    for(; i < times; i++) {
        CUDAKERNELCHECK((gpu_box_3d1r_kernel<<<grid_config, block_config, sm_size>>>(array_d[i % 2], array_d[(i + 1) % 2], input_h, rows, cols, lookup_table1_d, lookup_table2_d)));
    }
    CUDA_CHECK(hipDeviceSynchronize());

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "ConvStencil(3D): " << std::endl;
    std::cout << "Time = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "[ms]" << std::endl;
    
    double secs = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() / 1e6;
    std::cout << secs << std::endl;
    printf("GStencil/s = %f\n", ((double)input_m * input_n * input_h * times * 3) / secs / 1e9);
    
    CUDA_CHECK(hipMemcpy(out, array_d[i % 2], array_size, hipMemcpyDeviceToHost));

    return;
}

void gpu_box_3d1r_breakdown4(const double * __restrict__ in, double * __restrict__ out, const double * __restrict__ params, const int times, const int input_h, const int input_m, const int input_n) {
    double param_matrix_h[4][2][52 * 8] = {0.0};

    // Initialize parameter matrix
    for (int h = 0; h < 4; h++) {
        for (int col = 0; col < TENSOR_CORE_M; col++) {
            for(int i = 0; i < UNIT_LENGTH; i++) {
                for(int j = 0; j < UNIT_LENGTH; j++) {
                    if (j >= col) {
                        param_matrix_h[h][0][(i * UNIT_LENGTH + j) * 8 + col] = params[h * UNIT_LENGTH * UNIT_LENGTH + i * UNIT_LENGTH + j - col];
                    }
                }
            }
        }
        for (int col = 0; col < TENSOR_CORE_M; col++) {
            for(int i = 0; i < UNIT_LENGTH; i++) {
                for(int j = 0; j < UNIT_LENGTH; j++) {
                    if (j < col) {
                        param_matrix_h[h][1][(i * UNIT_LENGTH + j) * 8 + col] = params[h * UNIT_LENGTH * UNIT_LENGTH + i * UNIT_LENGTH + j - col + 7];
                    }
                }
            }
        }
    }

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(param_matrix_d), param_matrix_h, 4 * 2 * 8 * 52 * sizeof(double)));

    const int heights = input_h + 2 * HALO;
    const int rows = input_m + 2 * HALO;
    const int cols = input_n + 2 * HALO;
    const size_t array_size = heights * rows * cols * sizeof(double);
    double *array_d[2];
    CUDA_CHECK(hipMalloc(&array_d[0], array_size));
    CUDA_CHECK(hipMalloc(&array_d[1], array_size));
    CUDA_CHECK(hipMemset(array_d[0], 0, array_size));
    CUDA_CHECK(hipMemcpy(array_d[0], in, array_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(array_d[1], 0, array_size));

    const int BLOCK_M = (input_m + BLOCK_SIZE_ROW - 1) / BLOCK_SIZE_ROW; 
    const int BLOCK_N = (input_n + BLOCK_SIZE_COL - 1) / BLOCK_SIZE_COL; 
    dim3 grid_config(BLOCK_M, BLOCK_N);
    // dim3 grid_config(1, 1);
    dim3 block_config(32 * WARP_PER_BLOCK);
    int sm_size = (2 * SM_SIZE_ROW * SM_SIZE_COL + 19 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL) * sizeof(double);
    CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(breakdown4_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, sm_size));

    int lookup_table1_h[D_BLOCK_SIZE_ROW][D_BLOCK_SIZE_COL];
    int lookup_table2_h[D_BLOCK_SIZE_ROW][D_BLOCK_SIZE_COL];
    for (int i = 0; i < D_BLOCK_SIZE_ROW; i++) {
        for (int j = 0; j < D_BLOCK_SIZE_COL; j++) {
            if ((j + 1) % 8 != 0 && j < D_BLOCK_SIZE_COL - 2 * HALO - 1) {
                lookup_table1_h[i][j] = IDX2D(j / (UNIT_LENGTH + 1), UNIT_LENGTH * i + j % (UNIT_LENGTH + 1), SM_SIZE_COL);
            } else {
                lookup_table1_h[i][j] = SM_SIZE_ROW * SM_SIZE_COL - 1;
            }
            if ((j + 2) % 8 != 0 && j > 2 * HALO) {
                lookup_table2_h[i][j] = IDX2D((j - UNIT_LENGTH) / (UNIT_LENGTH + 1), UNIT_LENGTH * i + (j - UNIT_LENGTH) % (UNIT_LENGTH + 1), SM_SIZE_COL);
            } else {
                lookup_table2_h[i][j] = SM_SIZE_ROW * SM_SIZE_COL - 1;
            }
        }
    }

    int * lookup_table1_d;
    int * lookup_table2_d;
    CUDA_CHECK(hipMalloc(&lookup_table1_d, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int)));
    CUDA_CHECK(hipMalloc(&lookup_table2_d, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int)));
    CUDA_CHECK(hipMemcpy(lookup_table1_d, lookup_table1_h, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(lookup_table2_d, lookup_table2_h, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int), hipMemcpyHostToDevice));

    int i = 0;

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    for(; i < times; i++) {
        CUDAKERNELCHECK((breakdown4_kernel<<<grid_config, block_config, sm_size>>>(array_d[i % 2], array_d[(i + 1) % 2], input_h, rows, cols, lookup_table1_d, lookup_table2_d)));
    }
    CUDA_CHECK(hipDeviceSynchronize());

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "Experiment - Breakdown(3D) 4: " << std::endl;
    std::cout << "Time = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "[ms]" << std::endl;
    
    double secs = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() / 1e6;
    std::cout << secs << std::endl;
    printf("GStencil/s = %f\n\n", ((double)input_m * input_n * input_h * times * 3) / secs / 1e9);
    
    CUDA_CHECK(hipMemcpy(out, array_d[i % 2], array_size, hipMemcpyDeviceToHost));

    return;
}

void gpu_box_3d1r_breakdown3(const double * __restrict__ in, double * __restrict__ out, const double * __restrict__ params, const int times, const int input_h, const int input_m, const int input_n) {
    double param_matrix_h[4][2][52 * 8] = {0.0};

    // Initialize parameter matrix
    for (int h = 0; h < 4; h++) {
        for (int col = 0; col < TENSOR_CORE_M; col++) {
            for(int i = 0; i < UNIT_LENGTH; i++) {
                for(int j = 0; j < UNIT_LENGTH; j++) {
                    if (j >= col) {
                        param_matrix_h[h][0][(i * UNIT_LENGTH + j) * 8 + col] = params[h * UNIT_LENGTH * UNIT_LENGTH + i * UNIT_LENGTH + j - col];
                    }
                }
            }
        }
        for (int col = 0; col < TENSOR_CORE_M; col++) {
            for(int i = 0; i < UNIT_LENGTH; i++) {
                for(int j = 0; j < UNIT_LENGTH; j++) {
                    if (j < col) {
                        param_matrix_h[h][1][(i * UNIT_LENGTH + j) * 8 + col] = params[h * UNIT_LENGTH * UNIT_LENGTH + i * UNIT_LENGTH + j - col + 7];
                    }
                }
            }
        }
    }

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(param_matrix_d), param_matrix_h, 4 * 2 * 8 * 52 * sizeof(double)));

    const int heights = input_h + 2 * HALO;
    const int rows = input_m + 2 * HALO;
    const int cols = input_n + 2 * HALO;
    const size_t array_size = heights * rows * cols * sizeof(double);
    double *array_d[2];
    CUDA_CHECK(hipMalloc(&array_d[0], array_size));
    CUDA_CHECK(hipMalloc(&array_d[1], array_size));
    CUDA_CHECK(hipMemset(array_d[0], 0, array_size));
    CUDA_CHECK(hipMemcpy(array_d[0], in, array_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(array_d[1], 0, array_size));

    const int BLOCK_M = (input_m + BLOCK_SIZE_ROW - 1) / BLOCK_SIZE_ROW; 
    const int BLOCK_N = (input_n + BLOCK_SIZE_COL - 1) / BLOCK_SIZE_COL; 
    dim3 grid_config(BLOCK_M, BLOCK_N);
    // dim3 grid_config(1, 1);
    dim3 block_config(32 * WARP_PER_BLOCK);
    int sm_size = (2 * SM_SIZE_ROW * SM_SIZE_COL + 19 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL) * sizeof(double);
    CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(breakdown3_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, sm_size));

    int lookup_table1_h[D_BLOCK_SIZE_ROW][D_BLOCK_SIZE_COL];
    int lookup_table2_h[D_BLOCK_SIZE_ROW][D_BLOCK_SIZE_COL];
    for (int i = 0; i < D_BLOCK_SIZE_ROW; i++) {
        for (int j = 0; j < D_BLOCK_SIZE_COL; j++) {
            if ((j + 1) % 8 != 0 && j < D_BLOCK_SIZE_COL - 2 * HALO - 1) {
                lookup_table1_h[i][j] = IDX2D(j / (UNIT_LENGTH + 1), UNIT_LENGTH * i + j % (UNIT_LENGTH + 1), SM_SIZE_COL);
            } else {
                lookup_table1_h[i][j] = SM_SIZE_ROW * SM_SIZE_COL - 1;
            }
            if ((j + 2) % 8 != 0 && j > 2 * HALO) {
                lookup_table2_h[i][j] = IDX2D((j - UNIT_LENGTH) / (UNIT_LENGTH + 1), UNIT_LENGTH * i + (j - UNIT_LENGTH) % (UNIT_LENGTH + 1), SM_SIZE_COL);
            } else {
                lookup_table2_h[i][j] = SM_SIZE_ROW * SM_SIZE_COL - 1;
            }
        }
    }

    int * lookup_table1_d;
    int * lookup_table2_d;
    CUDA_CHECK(hipMalloc(&lookup_table1_d, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int)));
    CUDA_CHECK(hipMalloc(&lookup_table2_d, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int)));
    CUDA_CHECK(hipMemcpy(lookup_table1_d, lookup_table1_h, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(lookup_table2_d, lookup_table2_h, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int), hipMemcpyHostToDevice));

    int i = 0;

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    for(; i < times; i++) {
        CUDAKERNELCHECK((breakdown3_kernel<<<grid_config, block_config, sm_size>>>(array_d[i % 2], array_d[(i + 1) % 2], input_h, rows, cols, lookup_table1_d, lookup_table2_d)));
    }
    CUDA_CHECK(hipDeviceSynchronize());

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "Experiment - Breakdown(3D) 3: " << std::endl;
    std::cout << "Time = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "[ms]" << std::endl;
    
    double secs = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() / 1e6;
    std::cout << secs << std::endl;
    printf("GStencil/s = %f\n\n", ((double)input_m * input_n * input_h * times * 3) / secs / 1e9);
    
    CUDA_CHECK(hipMemcpy(out, array_d[i % 2], array_size, hipMemcpyDeviceToHost));

    return;
}

void gpu_box_3d1r_breakdown2(const double * __restrict__ in, double * __restrict__ out, const double * __restrict__ params, const int times, const int input_h, const int input_m, const int input_n) {

    double param_matrix_h[4][2][52 * 8] = {0.0};

    // Initialize parameter matrix
    for (int h = 0; h < 4; h++) {
        for (int col = 0; col < TENSOR_CORE_M; col++) {
            for(int i = 0; i < UNIT_LENGTH; i++) {
                for(int j = 0; j < UNIT_LENGTH; j++) {
                    if (j >= col) {
                        param_matrix_h[h][0][(i * UNIT_LENGTH + j) * 8 + col] = params[h * UNIT_LENGTH * UNIT_LENGTH + i * UNIT_LENGTH + j - col];
                    }
                }
            }
        }
        for (int col = 0; col < TENSOR_CORE_M; col++) {
            for(int i = 0; i < UNIT_LENGTH; i++) {
                for(int j = 0; j < UNIT_LENGTH; j++) {
                    if (j < col) {
                        param_matrix_h[h][1][(i * UNIT_LENGTH + j) * 8 + col] = params[h * UNIT_LENGTH * UNIT_LENGTH + i * UNIT_LENGTH + j - col + 7];
                    }
                }
            }
        }
    }

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(param_matrix_d), param_matrix_h, 4 * 2 * 8 * 52 * sizeof(double)));

    const int heights = input_h + 2 * HALO;
    const int rows = input_m + 2 * HALO;
    const int cols = input_n + 2 * HALO;
    const size_t array_size = heights * rows * cols * sizeof(double);
    double *array_d[2];
    CUDA_CHECK(hipMalloc(&array_d[0], array_size));
    CUDA_CHECK(hipMalloc(&array_d[1], array_size));
    CUDA_CHECK(hipMemset(array_d[0], 0, array_size));
    CUDA_CHECK(hipMemcpy(array_d[0], in, array_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(array_d[1], 0, array_size));

    const int BLOCK_M = (input_m + BLOCK_SIZE_ROW - 1) / BLOCK_SIZE_ROW; 
    const int BLOCK_N = (input_n + BLOCK_SIZE_COL - 1) / BLOCK_SIZE_COL; 
    dim3 grid_config(BLOCK_M, BLOCK_N);
    // dim3 grid_config(1, 1);
    dim3 block_config(32 * WARP_PER_BLOCK);
    int sm_size = (2 * SM_SIZE_ROW * SM_SIZE_COL + 19 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL) * sizeof(double);
    CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(breakdown2_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, sm_size));

    int lookup_table1_h[D_BLOCK_SIZE_ROW][D_BLOCK_SIZE_COL];
    int lookup_table2_h[D_BLOCK_SIZE_ROW][D_BLOCK_SIZE_COL];
    for (int i = 0; i < D_BLOCK_SIZE_ROW; i++) {
        for (int j = 0; j < D_BLOCK_SIZE_COL; j++) {
            if ((j + 1) % 8 != 0 && j < D_BLOCK_SIZE_COL - 2 * HALO - 1) {
                lookup_table1_h[i][j] = IDX2D(j / (UNIT_LENGTH + 1), UNIT_LENGTH * i + j % (UNIT_LENGTH + 1), SM_SIZE_COL);
            } else {
                lookup_table1_h[i][j] = SM_SIZE_ROW * SM_SIZE_COL - 1;
            }
            if ((j + 2) % 8 != 0 && j > 2 * HALO) {
                lookup_table2_h[i][j] = IDX2D((j - UNIT_LENGTH) / (UNIT_LENGTH + 1), UNIT_LENGTH * i + (j - UNIT_LENGTH) % (UNIT_LENGTH + 1), SM_SIZE_COL);
            } else {
                lookup_table2_h[i][j] = SM_SIZE_ROW * SM_SIZE_COL - 1;
            }
        }
    }

    int * lookup_table1_d;
    int * lookup_table2_d;
    CUDA_CHECK(hipMalloc(&lookup_table1_d, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int)));
    CUDA_CHECK(hipMalloc(&lookup_table2_d, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int)));
    CUDA_CHECK(hipMemcpy(lookup_table1_d, lookup_table1_h, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(lookup_table2_d, lookup_table2_h, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int), hipMemcpyHostToDevice));

    int i = 0;

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    for(; i < times; i++) {
        CUDAKERNELCHECK((breakdown2_kernel<<<grid_config, block_config, sm_size>>>(array_d[i % 2], array_d[(i + 1) % 2], input_h, rows, cols, lookup_table1_d, lookup_table2_d)));
    }
    CUDA_CHECK(hipDeviceSynchronize());

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "Experiment - Breakdown(3D) 2: " << std::endl;
    std::cout << "Time = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "[ms]" << std::endl;
    
    double secs = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() / 1e6;
    std::cout << secs << std::endl;
    printf("GStencil/s = %f\n\n", ((double)input_m * input_n * input_h * times * 3) / secs / 1e9);
    
    CUDA_CHECK(hipMemcpy(out, array_d[i % 2], array_size, hipMemcpyDeviceToHost));

    return;
}

void gpu_box_3d1r_breakdown1(const double * __restrict__ in, double * __restrict__ out, const double * __restrict__ params, const int times, const int input_h, const int input_m, const int input_n) {
    double param_matrix_h[4][2][52 * 8] = {0.0};

    // Initialize parameter matrix
    for (int h = 0; h < 4; h++) {
        for (int col = 0; col < TENSOR_CORE_M; col++) {
            for(int i = 0; i < UNIT_LENGTH; i++) {
                for(int j = 0; j < UNIT_LENGTH; j++) {
                    if (j >= col) {
                        param_matrix_h[h][0][(i * UNIT_LENGTH + j) * 8 + col] = params[h * UNIT_LENGTH * UNIT_LENGTH + i * UNIT_LENGTH + j - col];
                    }
                }
            }
        }
        for (int col = 0; col < TENSOR_CORE_M; col++) {
            for(int i = 0; i < UNIT_LENGTH; i++) {
                for(int j = 0; j < UNIT_LENGTH; j++) {
                    if (j < col) {
                        param_matrix_h[h][1][(i * UNIT_LENGTH + j) * 8 + col] = params[h * UNIT_LENGTH * UNIT_LENGTH + i * UNIT_LENGTH + j - col + 7];
                    }
                }
            }
        }
    }

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(param_matrix_d), param_matrix_h, 4 * 2 * 8 * 52 * sizeof(double)));

    const int heights = input_h + 2 * HALO;
    const int rows = input_m + 2 * HALO;
    const int cols = input_n + 2 * HALO;
    const size_t array_size = heights * rows * cols * sizeof(double);
    double *array_d[2];
    CUDA_CHECK(hipMalloc(&array_d[0], array_size));
    CUDA_CHECK(hipMalloc(&array_d[1], array_size));
    CUDA_CHECK(hipMemset(array_d[0], 0, array_size));
    CUDA_CHECK(hipMemcpy(array_d[0], in, array_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(array_d[1], 0, array_size));

    const int BLOCK_M = (input_m + BLOCK_SIZE_ROW - 1) / BLOCK_SIZE_ROW; 
    const int BLOCK_N = (input_n + BLOCK_SIZE_COL - 1) / BLOCK_SIZE_COL; 
    dim3 grid_config(BLOCK_M, BLOCK_N);
    // dim3 grid_config(1, 1);
    dim3 block_config(32 * WARP_PER_BLOCK);
    int sm_size = (2 * SM_SIZE_ROW * SM_SIZE_COL + 19 * BLOCK_SIZE_ROW * BLOCK_SIZE_COL) * sizeof(double);
    CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(breakdown1_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, sm_size));

    int lookup_table1_h[D_BLOCK_SIZE_ROW][D_BLOCK_SIZE_COL];
    int lookup_table2_h[D_BLOCK_SIZE_ROW][D_BLOCK_SIZE_COL];
    for (int i = 0; i < D_BLOCK_SIZE_ROW; i++) {
        for (int j = 0; j < D_BLOCK_SIZE_COL; j++) {
            if ((j + 1) % 8 != 0 && j < D_BLOCK_SIZE_COL - 2 * HALO - 1) {
                lookup_table1_h[i][j] = IDX2D(j / (UNIT_LENGTH + 1), UNIT_LENGTH * i + j % (UNIT_LENGTH + 1), SM_SIZE_COL);
            } else {
                lookup_table1_h[i][j] = SM_SIZE_ROW * SM_SIZE_COL - 1;
            }
            if ((j + 2) % 8 != 0 && j > 2 * HALO) {
                lookup_table2_h[i][j] = IDX2D((j - UNIT_LENGTH) / (UNIT_LENGTH + 1), UNIT_LENGTH * i + (j - UNIT_LENGTH) % (UNIT_LENGTH + 1), SM_SIZE_COL);
            } else {
                lookup_table2_h[i][j] = SM_SIZE_ROW * SM_SIZE_COL - 1;
            }
        }
    }

    int * lookup_table1_d;
    int * lookup_table2_d;
    CUDA_CHECK(hipMalloc(&lookup_table1_d, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int)));
    CUDA_CHECK(hipMalloc(&lookup_table2_d, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int)));
    CUDA_CHECK(hipMemcpy(lookup_table1_d, lookup_table1_h, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(lookup_table2_d, lookup_table2_h, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int), hipMemcpyHostToDevice));

    int i = 0;

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    for(; i < times; i++) {
        CUDAKERNELCHECK((breakdown1_kernel<<<grid_config, block_config, sm_size>>>(array_d[i % 2], array_d[(i + 1) % 2], input_h, rows, cols, lookup_table1_d, lookup_table2_d)));
    }
    CUDA_CHECK(hipDeviceSynchronize());

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "Experiment - Breakdown(3D) 1: " << std::endl;
    std::cout << "Time = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "[ms]" << std::endl;
    
    double secs = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() / 1e6;
    std::cout << secs << std::endl;
    printf("GStencil/s = %f\n\n", ((double)input_m * input_n * input_h * times * 3) / secs / 1e9);
    
    CUDA_CHECK(hipMemcpy(out, array_d[i % 2], array_size, hipMemcpyDeviceToHost));

    return;
}